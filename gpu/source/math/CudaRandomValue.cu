/*
 * Copyright Regents of the University of Minnesota, 2015.  This software is released under the following license: http://opensource.org/licenses/GPL-2.0
 * Source code originally developed at the University of Minnesota Interactive Visualization Lab (http://ivlab.cs.umn.edu).
 *
 * Code author(s):
 * 		Dan Orban (dtorban)
 */

#include <PFGpu/math/CudaRandomValue.cuh>

namespace PFCore {
namespace math {

CudaRandomValue::CudaRandomValue(int deviceId, int size) : RandomArrayValue(), _deviceId(deviceId) {
	numRand = size;
	hipSetDevice(_deviceId);
	hipMalloc(&rnd, numRand*sizeof(float));

	float* rndLocal = new float[size];
	for (int f = 0; f < numRand; f++)
	{
		rndLocal[f] = float(std::rand())/RAND_MAX;	
	}

	hipMemcpy(rnd, rndLocal, numRand*sizeof(float), hipMemcpyHostToDevice);

	delete[] rndLocal;
}

CudaRandomValue::~CudaRandomValue() {
	hipSetDevice(_deviceId);
	hipFree(rnd);
}

/*extern "C"
RandomValue* createCudaRandomValue(int deviceId, int size)
{
	return new CudaRandomValue(deviceId, size);
}*/

} /* namespace math */
} /* namespace PFCore */
