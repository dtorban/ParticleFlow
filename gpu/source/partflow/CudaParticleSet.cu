/*
 * Copyright Regents of the University of Minnesota, 2015.  This software is released under the following license: http://opensource.org/licenses/GPL-2.0
 * Source code originally developed at the University of Minnesota Interactive Visualization Lab (http://ivlab.cs.umn.edu).
 *
 * Code author(s):
 * 		Dan Orban (dtorban)
 */

#include <gpu/include/PFGpu/partflow/CudaParticleSet.cuh>

namespace PFCore {
namespace partflow {

CudaParticleSet::CudaParticleSet(int deviceId, int numParticles, int numValues, int numVectors) : ParticleSet(), _deviceId(deviceId) {
	_numParticles = numParticles;
	_numValues = numValues;
	_numVectors = numVectors;
	hipSetDevice(deviceId);
	hipMalloc(&_positions, numParticles*sizeof(math::vec3));
	hipMalloc(&_values, numParticles*numValues*sizeof(float));
	hipMalloc(&_vectors, numParticles*numVectors*sizeof(math::vec3));
}

CudaParticleSet::~CudaParticleSet() {
	hipSetDevice(_deviceId);
	hipFree(_positions);
	hipFree(_values);
	hipFree(_vectors);
}

} /* namespace partflow */
}
