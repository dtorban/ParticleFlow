/*
 * Copyright Regents of the University of Minnesota, 2015.  This software is released under the following license: http://opensource.org/licenses/GPL-2.0
 * Source code originally developed at the University of Minnesota Interactive Visualization Lab (http://ivlab.cs.umn.edu).
 *
 * Code author(s):
 * 		Dan Orban (dtorban)
 */

#include <gpu/include/PFGpu/partflow/CudaParticleSet.cuh>

namespace PFCore {
namespace partflow {

CudaParticleSet::CudaParticleSet(int numParticles, int numValues, int numVectors, int numSteps) : ParticleSet(numParticles, numValues, numVectors, numSteps), _deviceId(-1)
{
}

CudaParticleSet::CudaParticleSet(int deviceId, int numParticles, int numValues, int numVectors, int numSteps) : ParticleSet(), _deviceId(deviceId) {
	_numParticles = numParticles;
	_numValues = numValues;
	_numVectors = numVectors;
	_numSteps = numSteps;
	hipSetDevice(deviceId);
	hipMalloc(&_positions, numSteps*numParticles*sizeof(math::vec3));
	hipMalloc(&_values, numSteps*numParticles*numValues*sizeof(float));
	hipMalloc(&_vectors, numSteps*numParticles*numVectors*sizeof(math::vec3));
}

CudaParticleSet::~CudaParticleSet() {
	if (_deviceId >= 0)
	{
		hipSetDevice(_deviceId);
		hipFree(_positions);
		hipFree(_values);
		hipFree(_vectors);
	}
}

void CudaParticleSet::copy(const ParticleSetView& particleSet, void* dst, const void* src, size_t size)
{
	// Both local
	if (getDeviceId() < 0 && particleSet.getDeviceId() < 0)
	{
		ParticleSet::copy(particleSet, dst, src, size);
	}
	// On same device
	else if (getDeviceId() == particleSet.getDeviceId())
	{
		// TODO: kernal to copy to same device
		hipMemcpy(dst, src, size, hipMemcpyDeviceToDevice);
	}
	// One of the devices is host
	else if (getDeviceId() < 0 || particleSet.getDeviceId() < 0)
	{
		hipMemcpy(dst, src, size, getDeviceId() < 0 ? hipMemcpyDeviceToHost :  hipMemcpyHostToDevice);
	}
	// Peer to peer
	else
	{
		hipMemcpyPeer(dst, getDeviceId(), src, particleSet.getDeviceId(), size);
	}
}

extern "C"
ParticleSet* createCudaParticleSet(int deviceId, int numParticles, int numValues, int numVectors, int numSteps)
{
	if (deviceId >= 0)
	{
		return new CudaParticleSet(deviceId, numParticles, numValues, numVectors, numSteps);
	}
	else
	{
		return new CudaParticleSet(numParticles, numValues, numVectors, numSteps);
	}
}

} /* namespace partflow */
}
