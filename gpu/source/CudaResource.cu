/*
 * CudaResource.cpp
 *
 *  Created on: Aug 16, 2015
 *      Author: dtorban
 */

#if defined(WIN32)
#define NOMINMAX
#include <windows.h>
#endif

#include <PFGpu/CudaResource.cuh>
#include <cuda_gl_interop.h>

namespace PFCore {

extern "C"
GpuResource* gpuRegisterResource(int deviceId, int resourceId)
{
	return new CudaResource(deviceId, resourceId);
}

CudaResource::CudaResource(int deviceId, int resourceId) : _deviceId(deviceId), _resourceId(resourceId) {
	hipSetDevice(_deviceId);
	cudaGLSetGLDevice(_deviceId);
	hipGraphicsGLRegisterBuffer(&resource, resourceId, cudaGraphicsMapFlagsNone);
}

CudaResource::~CudaResource() {
	hipSetDevice(_deviceId);
	cudaGLSetGLDevice(_deviceId);
	hipGraphicsUnregisterResource(resource);
}

bool CudaResource::map()
{
	hipSetDevice(_deviceId);
	cudaGLSetGLDevice(_deviceId);
	hipGraphicsMapResources(1, &resource);
	return true;
}

int CudaResource::getData(void** data)
{
	hipSetDevice(_deviceId);
	cudaGLSetGLDevice(_deviceId);
	size_t size;
	hipGraphicsResourceGetMappedPointer(data, &size, resource);
	return size;
}

void CudaResource::unmap()
{
	hipSetDevice(_deviceId);
	cudaGLSetGLDevice(_deviceId);
	hipGraphicsUnmapResources(1, &resource);
}

int CudaResource::getDeviceId()
{
	return _deviceId;
}

} /* namespace PFCore */
