/*
 * CudaResource.cpp
 *
 *  Created on: Aug 16, 2015
 *      Author: dtorban
 */

#include <PFGpu/CudaResource.cuh>
#include <cuda_gl_interop.h>

namespace PFCore {

extern "C"
GpuResource* gpuRegisterResource(int deviceId, int resourceId)
{
	return new CudaResource(deviceId, resourceId);
}

CudaResource::CudaResource(int deviceId, int resourceId) : _deviceId(deviceId), _resourceId(resourceId) {
	hipSetDevice(_deviceId);
	cudaGLSetGLDevice(_deviceId);
	hipGraphicsGLRegisterBuffer(&resource, resourceId, cudaGraphicsMapFlagsNone);
}

CudaResource::~CudaResource() {
	hipSetDevice(_deviceId);
	cudaGLSetGLDevice(_deviceId);
	hipGraphicsUnregisterResource(resource);
}

void CudaResource::map(void*& data)
{
	hipSetDevice(_deviceId);
	cudaGLSetGLDevice(_deviceId);
	hipGraphicsMapResources(1, &resource);
	size_t size;
	hipGraphicsResourceGetMappedPointer((void **)(&data), &size, resource);
}

void CudaResource::unmap()
{
	hipSetDevice(_deviceId);
	cudaGLSetGLDevice(_deviceId);
	hipGraphicsUnmapResources(1, &resource);
}

} /* namespace PFCore */
